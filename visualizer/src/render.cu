#include "hip/hip_runtime.h"
#include <format>
#include <iostream>
#include <tuple>

#include <openvdb/openvdb.h>
#include <nanovdb/tools/CreateNanoGrid.h>
#include <nanovdb/util/IO.h>
#include <nanovdb/cuda/DeviceBuffer.h>

#include "glad/glad.h"
#include <GLFW/glfw3.h>
#include <GL/gl.h>
#include <cuda_gl_interop.h>

#include "generateImage.cuh"


using Vec3f = typename nanovdb::math::Vec3f;

struct CameraData {
    uint32_t mImgHeight; // in pixels
    uint32_t mImgWidth; // in pixels
    uint32_t mImgChannels = 3 ; // 3 channels for RGB

    // extrinsics
    Vec3f mPosition;
    Vec3f mDirection;
    Vec3f mUp;
    Vec3f mRight;
    float mYaw;
    float mPitch;

    // intrinsics
    float mFOV = 90.0f;
    float mFocalLength = 1.0f;
    float mHlfViewPlaneWidth;
    float mFarClip = 500.0f;

    float mMovementSpeed = 1.0f;
    float mSwivelSensitivity = 0.07f;

    CameraData() = delete;

    CameraData(Vec3f position, Vec3f target, uint32_t imgWidth, uint32_t imgHeight) {
        mPosition = position;
        mImgWidth = imgWidth;
        mImgHeight = imgHeight;
        mDirection = (target - position).normalize();
        mHlfViewPlaneWidth = tan((mFOV / 2) * nanovdb::math::pi<float>() / 180.0f) * mFocalLength;
        mRight = Vec3f{0, 1, 0}.cross(mDirection).normalize();
        mUp = mDirection.cross(mRight);

        mYaw = std::atan2(mDirection[2], mDirection[0]) * 180.0f / nanovdb::math::pi<float>();
        mPitch = std::asin(mDirection[1]) * 180.0f / nanovdb::math::pi<float>();
    }

    void computeDirection(){
        const float conversionFactor = nanovdb::math::pi<float>() / 180.0f;
        float radYaw = mYaw * conversionFactor;
        float radPitch = mPitch * conversionFactor;

        mDirection[0] = cos(radYaw) * cos(radPitch);
        mDirection[1] = sin(radPitch);
        mDirection[2] = sin(radYaw) * cos(radPitch);

        mRight = Vec3f{0, 1, 0}.cross(mDirection).normalize();
        mUp = mDirection.cross(mRight);
    }
};

struct WindowUserData {
    CameraData* cameraData;
    float lastX = 512.0f; // used for mouseCallback
    float lastY = 512.0f; // used for mouseCallback
    int initalFrame = 0; // used to prevent view snapping on startup in mouseCallback

    WindowUserData(CameraData* pCameraData) : cameraData(pCameraData) {}
};

std::tuple<GLFWwindow*, GLuint, GLuint, GLuint, GLuint> GLInit(WindowUserData* windowUserData);
std::tuple<GLuint, hipGraphicsResource_t, hipSurfaceObject_t> GLCUDAInit(hipStream_t stream, uint32_t imgWidth, uint32_t imgHeight);
void displayUpdate(GLFWwindow* window, GLuint texture, GLuint64 VAO, GLuint shaderProgram);
void processInput(GLFWwindow* window);
void mouseCallback(GLFWwindow* window, double xpos, double ypos);
uint32_t getDefaultShaderProgram();
void frameBufferSizeCallback(GLFWwindow* window, int width, int height);

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <path_to_nvdb_file>" << std::endl;
        return 1;
    }

    std::string nvdbPath = argv[1];
    auto handle = nanovdb::io::readGrid<nanovdb::cuda::DeviceBuffer>(nvdbPath);

    hipStream_t stream; // create a CUDA stream to allow for asynchronous copy of pinned CUDA memory
    hipStreamCreate(&stream);

    handle.deviceUpload(stream, false); // Copy the NanoVDB grid to the GPU asynchronously

    auto* cpuGrid = handle.grid<int32_t>(); // get a (raw) pointer to the grid of type int32 on the CPU
    auto* deviceGrid = handle.deviceGrid<int32_t>(); // get a (raw) pointer to the grid of type int32 on the GPU

    if (!deviceGrid || !cpuGrid)
        throw std::runtime_error("GridHandle did not contain a grid with the value type Int32");

    Vec3f position = {0, 200, 200};
    Vec3f target = {100, 200, 200};
    CameraData cameraData(position, target, 1024, 1024);
    WindowUserData windowUserData(&cameraData);

    auto [window, shaderProgram, VAO, VBO, EBO] = GLInit(&windowUserData);
    if (!window) return 1;
    auto [glTexture, cuda_resource, cuda_surface_write] = GLCUDAInit(stream, cameraData.mImgWidth, cameraData.mImgHeight);

    while (!glfwWindowShouldClose(window))
    {
        processInput(window);

        hipGraphicsMapResources(1, &cuda_resource, stream);

        dim3 blockSize(16, 16); // 16*16 = 256 threads
        dim3 gridSize(((cameraData.mImgWidth + blockSize.x - 1)/blockSize.x), ((cameraData.mImgHeight + blockSize.y - 1)/blockSize.y));
        GLCUDAWriteToTex<<<gridSize, blockSize>>>(deviceGrid, cuda_surface_write, cameraData.mImgWidth, cameraData.mImgHeight, cameraData.mDirection,
                                                  cameraData.mRight, cameraData.mUp, cameraData.mPosition, cameraData.mFocalLength, cameraData.mHlfViewPlaneWidth);
        hipDeviceSynchronize();

        hipGraphicsUnmapResources(1, &cuda_resource, stream);

        displayUpdate(window, glTexture, VAO, shaderProgram);

    }

    hipGraphicsUnregisterResource(cuda_resource);
    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &EBO);
    glDeleteTextures(1, &glTexture);
    glfwTerminate();

    return 0;
}

std::tuple<GLFWwindow*, GLuint, GLuint, GLuint, GLuint> GLInit(WindowUserData* windowUserData) {
    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return {nullptr, NULL, NULL, NULL, NULL};
    }

    GLFWwindow* window = glfwCreateWindow(1024, 1024, "NanoVDB Renderer", NULL, NULL);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return {nullptr, NULL, NULL, NULL, NULL};
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
        std::cerr << "Failed to initialize GLAD" << std::endl;
        glfwTerminate();
        return {nullptr, NULL, NULL, NULL, NULL};
    }


    // Check that opengl is using nvidia gpu for cuda interop (May use integrated graphics instead by default)
    const char* renderer = (const char*)glGetString(GL_RENDERER);
    const char* vendor = (const char*)glGetString(GL_VENDOR);

    if (strstr(vendor, "NVIDIA") == NULL) {
        std::cerr << "Error: OpenGL Not using NVIDIA GPU. Run with '__NV_PRIME_RENDER_OFFLOAD=1 __GLX_VENDOR_LIBRARY_NAME=nvidia'. (linux)" << std::endl;
        std::cout << "Renderer: " << renderer << std::endl << "Vendor: " << vendor << std::endl;
        glfwTerminate();
        return {nullptr, NULL, NULL, NULL, NULL};
    }

    glfwSetFramebufferSizeCallback(window, frameBufferSizeCallback);
    glfwSetCursorPosCallback(window, mouseCallback);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

    glfwSetWindowUserPointer(window, windowUserData);

////////////////////////        finished glfw setup        /////////////////////////////////////////////

    GLfloat vertices[] =
    {
        -1.0f, -1.0f, 0.0f,  0.0f, 0.0f,  // bottom left
        -1.0f,  1.0f, 0.0f,  0.0f, 1.0f,  // top left
         1.0f,  1.0f, 0.0f,  1.0f, 1.0f,  // top right
         1.0f, -1.0f, 0.0f,  1.0f, 0.0f   // bottom right
    };

    // Indices for vertices order
    GLuint indices[] =
    {
        0, 2, 1, // upper left triangle
        0, 3, 2, // lower right triangle
    };

    unsigned int shaderProgram = getDefaultShaderProgram();

    unsigned int VAO;
    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);

    // Create a vertex buffer object (VBO)
    unsigned int VBO;
    glGenBuffers(1, &VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

    // Create an element buffer object (EBO)
    unsigned int EBO;
    glGenBuffers(1, &EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indices), indices, GL_STATIC_DRAW);

    // Update the vertex attribute pointers for the position (location 0)
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);

    // Add the vertex attribute pointer for the texture coordinates (location 1)
    glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(3 * sizeof(float)));
    glEnableVertexAttribArray(1);

    // Unbind the VAO
    glBindVertexArray(0);

    return {window, shaderProgram, VAO, VBO, EBO};
}

std::tuple<GLuint, hipGraphicsResource_t, hipSurfaceObject_t> GLCUDAInit(hipStream_t stream, uint32_t imgWidth, uint32_t imgHeight)
{
    hipSetDevice(0);

    GLuint glTexture;
    glGenTextures(1, &glTexture);
    glBindTexture(GL_TEXTURE_2D, glTexture);
    // Set texture parameters
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    // Reserve space in GPU memory
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, imgWidth, imgHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);
    // Unbind texture
    glBindTexture(GL_TEXTURE_2D, 0);

    //CUDA
    hipGraphicsResource_t cuda_resource;
    hipGraphicsGLRegisterImage(&cuda_resource, glTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    hipGraphicsMapResources(1, &cuda_resource, stream);

    hipArray* glCudaArray;
    hipGraphicsSubResourceGetMappedArray(&glCudaArray, cuda_resource, 0, 0);
    hipGraphicsUnmapResources(1, &cuda_resource, stream);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = glCudaArray;

    hipSurfaceObject_t cuda_surface_write;
    hipCreateSurfaceObject(&cuda_surface_write, (const hipResourceDesc *)&resDesc);

    return {glTexture, cuda_resource, cuda_surface_write};
}

void displayUpdate(GLFWwindow* window, GLuint texture, GLuint64 VAO, GLuint shaderProgram) {

    glClearColor(0.2, 0.5, 1, 1);
    glClear(GL_COLOR_BUFFER_BIT);

    // Draw our square
    glUseProgram(shaderProgram);
    glBindTexture(GL_TEXTURE_2D, texture);
    glBindVertexArray(VAO);
    glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
    
    glBindVertexArray(0);
    glBindTexture(GL_TEXTURE_2D, 0);

    glfwSwapBuffers(window);
    glfwPollEvents();
}

void processInput(GLFWwindow* window)
{
    WindowUserData* windowUserData = (WindowUserData*)glfwGetWindowUserPointer(window);
    CameraData* camera = windowUserData->cameraData;

    if(glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);

    else if(glfwGetKey(window, GLFW_KEY_W)) {
        camera->mPosition += camera->mDirection * camera->mMovementSpeed;
    } else if(glfwGetKey(window, GLFW_KEY_S)) {
        camera->mPosition -= camera->mDirection * camera->mMovementSpeed;
    }
    
    if(glfwGetKey(window, GLFW_KEY_A)) {
        camera->mPosition -= camera->mRight * camera->mMovementSpeed;
    }  else if(glfwGetKey(window, GLFW_KEY_D)) {
        camera->mPosition += camera->mRight * camera->mMovementSpeed;
    }

}

void mouseCallback(GLFWwindow* window, double xPos, double yPos)
{
    WindowUserData* windowUserData = (WindowUserData*)glfwGetWindowUserPointer(window);
    CameraData* camera = windowUserData->cameraData;

    // prevent orientation snapping on startup
    if (windowUserData->initalFrame < 5) {
        windowUserData->lastX = xPos;
        windowUserData->lastY = yPos;
        windowUserData->initalFrame += 1;
        return;
    }
    
    float xOffset = xPos - windowUserData->lastX;
    float yOffset = yPos - windowUserData->lastY;

    windowUserData->lastX = xPos;
    windowUserData->lastY = yPos;

    xOffset *= camera->mSwivelSensitivity;
    yOffset *= camera->mSwivelSensitivity;

    camera->mYaw -= xOffset;
    camera->mPitch -= yOffset;

    if (camera->mPitch > 89.0f)
        camera->mPitch = 89.0f;
    else if (camera->mPitch < -89.0f)
        camera->mPitch = -89.0f;

    camera->computeDirection();
}

void frameBufferSizeCallback(GLFWwindow* window, int width, int height) {
    glViewport(0, 0, width, height);
}

uint32_t getDefaultShaderProgram() {
    const char *vertexShaderSource =
    "#version 330 core\n"
    "layout (location = 0) in vec3 aPos;\n"
    "layout (location = 1) in vec2 aTexCoord;\n"
    "out vec2 TexCoord;\n"
    "void main()\n"
    "{\n"
    "   gl_Position = vec4(aPos.x, aPos.y, aPos.z, 1.0);\n"
    "   TexCoord = aTexCoord;\n"
    "}\0";

const char *fragmentShaderSource =
    "#version 330 core\n"
    "out vec4 FragColor;\n"
    "in vec2 TexCoord;\n"
    "uniform sampler2D texture1;\n"
    "void main()\n"
    "{\n"
    "   FragColor = texture(texture1, TexCoord);\n"
    "}\n\0";

    unsigned int vertexShader = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(vertexShader, 1, &vertexShaderSource, NULL);
    glCompileShader(vertexShader);

    // fragment shader
    unsigned int fragmentShader = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(fragmentShader, 1, &fragmentShaderSource, NULL);
    glCompileShader(fragmentShader);

    unsigned int shaderProgram = glCreateProgram();
    glAttachShader(shaderProgram, vertexShader);
    glAttachShader(shaderProgram, fragmentShader);
    glLinkProgram(shaderProgram);

    glDeleteShader(vertexShader);
    glDeleteShader(fragmentShader);

    return shaderProgram;
}