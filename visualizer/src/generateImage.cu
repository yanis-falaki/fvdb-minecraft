#include "hip/hip_runtime.h"
#include <nanovdb/NanoVDB.h>
#include <nanovdb/math/Math.h>
#include <nanovdb/math/Ray.h>
#include <nanovdb/math/HDDA.h>

#include "glad/glad.h"
#include <GLFW/glfw3.h>
#include <GL/gl.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#include "generateImage.cuh"

using Vec3f = nanovdb::math::Vec3f;

template<typename RayT, typename AccT>
inline __hostdev__ bool firstActiveCustom(RayT& ray, AccT& acc, nanovdb::math::Coord &ijk, float& t)
{
    if (!ray.clip(acc.root().bbox()) || ray.t1() > 1e20) {// clip ray to bbox
        return false;// missed or undefined bbox
    }
    static const float Delta = 0.001f;// forward step-size along the ray to avoid getting stuck
    ray.setMinTime(ray.t0() - 0.0001f);// step back a small delta to avoid missing a first voxel at the edge of the bbox
    t = ray.t0();// initiate time
    ijk = RoundDown<nanovdb::math::Coord>(ray.start()); // first voxel inside bbox
    for (nanovdb::math::HDDA<RayT, nanovdb::math::Coord> hdda(ray, acc.getDim(ijk, ray)); !acc.isActive(ijk); hdda.update(ray, acc.getDim(ijk, ray))) {
        if (!hdda.step()) return false;// leap-frog HDDA and exit if ray bound is exceeded
        t = hdda.time() + Delta;// update time
        ijk = RoundDown<nanovdb::math::Coord>( ray(t) );// update ijk
    }
    return true;
}

__global__ void GLCUDAWriteToTex(nanovdb::Int32Grid* grid, hipSurfaceObject_t cuda_surface_write,
                    size_t imgWidth, size_t imgHeight,
                    Vec3f dirVec, Vec3f rightVec, Vec3f upVec,
                    Vec3f posVec, float focalLength, float hlfViewPlaneWidth) {
    
    // Calculate row and column ids, as well as element offset
    uint32_t rowId = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate column id and pixel pointer
    uint32_t colId = blockIdx.x * blockDim.x + threadIdx.x;

    if (rowId >= imgHeight || colId >= imgWidth) {
        return;
    }

    // Map from pixel coordinates to screen space coords, calculate rayDirection
    float screen_i = (((float)colId / imgWidth) - 0.5f) * hlfViewPlaneWidth; // horizontal displacement
    float screen_j = (((float)rowId / imgHeight) - 0.5f) * hlfViewPlaneWidth; // vertical displacement
    Vec3f rayDir = (dirVec * focalLength + rightVec * screen_i + upVec * screen_j).normalize();

    // Create Ray to be used for tracing
    float t;
    float startTime = 0.01f;
    float endTime = 500.0f;
    nanovdb::math::Ray<float> ray(posVec, rayDir, startTime, endTime);

    // Trace Ray
    auto acc = grid->getAccessor();
    nanovdb::math::Coord coord;
    //bool hit = nanovdb::math::firstActive(ray, acc, coord, t); causes artifacts
    bool hit = firstActiveCustom(ray, acc, coord, t);

    if (!hit) {
        surf2Dwrite(make_uchar4(128, 128, 128, 255), cuda_surface_write, colId * sizeof(uchar4), rowId);
    } else {
        int value = acc.getValue(coord);
        surf2Dwrite(make_uchar4((value * 35) % 256, (value * 28) % 256, (value * 15) % 256, 255), cuda_surface_write, colId * sizeof(uchar4), rowId);
    }
}